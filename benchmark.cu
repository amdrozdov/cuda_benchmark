#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cassert>
#include "benchmark.h"

// Runs vectors sum benchmark for given vector size
std::pair<int, int> sum_benchmark(int size, int grid_d, int block_d) {
    std::cout << "Add bench: " << size << " elems" << ". Grid=" << grid_d << "x";
    std::cout << block_d << std::endl;

    auto a = create_vector(size);
    auto b = create_vector(size);
    auto res1 = create_vector(size);
    auto res2 = create_vector(size);

    // Create random vectors of given size
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distr(0.1, 1000.0);
    for (int i = 0; i < size; i++) {
        a[i] = distr(gen);
        b[i] = distr(gen);
    }

    // Perform addition
    auto gpu_time = gpu_add(a, b, res1, size, grid_d, block_d);
    auto cpu_time = cpu_add(a, b, res2, size);

    // Check that all blocks are calculated correctly
    for (int i = 0; i < size; i++) {
        assert(res2[i]== res1[i]);
    }

    std::cout << "CPU time " << cpu_time << " mcs" << std::endl;
    std::cout << "GPU time " << gpu_time << " mcs" << std::endl;
    std::cout << "Ratio: " << float(cpu_time) / float(gpu_time) << std::endl << std::endl;

    // Cleanup memory
    free(a);
    free(b);
    free(res1);
    free(res2);
    return std::pair(gpu_time, cpu_time);
}

// Runs matrix multiplication benchmark for given matrix N*N size
std::pair<int,int> matrix_benchmark(int size){
    std::cout << "Matrix bench: " << size << std::endl;
    float** a = create_matrix(size);
    float** b = create_matrix(size);

    // Create random matrix N*N
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> distr(1.0, 3.0);
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            a[i][j] = float(round(distr(gen)));
            b[i][j] = float(round(distr(gen)));
        }
    }

    // multiply C=A*B
    float** res1 = create_matrix(size);
    float** res2 = create_matrix(size);
    auto cpu_time = cpu_sq_matrix_mul(a, b, res1, size);
    auto gpu_time = gpu_sq_matrix_mul(a, b, res2, size);

    // validate result
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            assert(res1[i][j] == res2[i][j]);
        }
    }

    std::cout << "CPU time " << cpu_time << " mcs" << std::endl;
    std::cout << "GPU time " << gpu_time << " mcs" << std::endl;
    std::cout << "Ratio: " << float(cpu_time) / float(gpu_time) << std::endl << std::endl;

    // Clean memory
    free(a);
    free(b);
    free(res1);
    free(res2);

    return std::pair(gpu_time, cpu_time);
}

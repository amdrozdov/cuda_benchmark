#include "hip/hip_runtime.h"
#include<iostream>
#include<chrono>
#include "compute.h"

#define assert_gpu_error(ans) { gpu_assert((ans), __FILE__, __LINE__); }

// Raises assertion in case of GPU errors (memory/operation errors)
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPU assert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Naive CPU vector sum
u_int64_t cpu_add(float* a, float* b, float*c, int size) {
    auto start = std::chrono::system_clock::now();

    for(int i =0;i<size;i++){
        c[i] = a[i] + b[i];
    }
    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(
            end - start);
    return elapsed.count();
}

// GPU vector sum kernel
__global__ void add(float* a, float* b, float* c) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

// Wrapper for GPU sum kernel (handles cuda memory allocation/transfer)
u_int64_t gpu_add(float* a, float *b, float* res, int size, int grid_d, int block_d){
    size_t total_bytes = size * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, total_bytes);
    hipMalloc(&d_B, total_bytes);
    hipMalloc(&d_C, total_bytes);

    hipMemcpy(d_A, a, total_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, total_bytes, hipMemcpyHostToDevice);

    auto start = std::chrono::system_clock::now();

    add<<<grid_d, block_d>>>(d_A, d_B, d_C);

    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(
            end - start);

    hipMemcpy(res, d_C, total_bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    assert_gpu_error( hipPeekAtLastError() );
    assert_gpu_error( hipDeviceSynchronize() );
    return elapsed.count();
}

// GPU matrix multiplication
__global__ void matrix_mul(float* a, float* b, float* c, int size){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= size || y >= size){
        return;
    }

    float result = 0;
    for(int i =0;i<size;i++){
        result += a[y*size + i] * b[i*size + x];
    }
    c[y*size + x] = result;
}

// GPU matrix multiplication wrapper (handles cuda memory allocation/transfer)
u_int64_t gpu_sq_matrix_mul(float** a, float** b, float** c, int size){
    // For matrix we will use 32 elements blocks in the grid
    int block_size = 32;
    // Calculate blocks/grid size for parallel execution on GPU
    dim3 grid(ceil(size/float(block_size)), ceil(size/float(block_size)));
    dim3 block(block_size, block_size, 1);

    // Convert to 2d matrixes into 1d array of floats
    size_t total_bytes = sizeof(float) * size * size;
    float *c_a = convert_to_1d(a, size);
    float *c_b = convert_to_1d(b, size);
    float *c_c = convert_to_1d(c, size);

    // Allocate cuda memory for all operands
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, total_bytes);
    hipMalloc(&d_B, total_bytes);
    hipMalloc(&d_C, total_bytes);
    // Transfer data to VRAM
    hipMemcpy(d_A, c_a, total_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, c_b, total_bytes, hipMemcpyHostToDevice);

    // Run the multiplication kernel on GPU using transferred data
    // This call is wrapped in the std::chrono time measurement
    auto start = std::chrono::system_clock::now();
    matrix_mul<<<grid, block>>>(d_A, d_B, d_C, size);
    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(
            end - start);

    // Transfer the multiplication result back from VRAM to regular memory
    hipMemcpy(c_c, d_C, total_bytes, hipMemcpyDeviceToHost);

    // Unpack 1d array into 2d matrix
    int i=0;
    int j=0;
    for(int x=0;x<size*size;x++){
        c[i][j] = c_c[x];
        j++;
        if(j >= size){
            j = 0;
            i++;
        }
    }

    // Cleanup VRAM and RAM
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(c_a);
    free(c_b);
    free(c_c);

    // Check GPU errors/assert
    assert_gpu_error( hipPeekAtLastError() );
    assert_gpu_error( hipDeviceSynchronize() );

    // Return calculation time, + float** c contains the result
    return elapsed.count();
}

// Naive CPU matrix muliplication
u_int64_t  cpu_sq_matrix_mul(float** a, float** b, float** c, int size){
    auto start = std::chrono::system_clock::now();
    for(int i=0;i<size;i++){
        for(int j=0;j<size;j++){
            for(int k=0;k<size;k++) {
                c[i][j] += a[i][k] * b[k][j];
            }
        }
    }
    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(
            end - start);
    return elapsed.count();
}
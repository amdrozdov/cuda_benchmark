#include "hip/hip_runtime.h"
#include<vector>
#include <iostream>
#include <fstream>
#include "benchmark.h"

void sum_bench(){
    std::vector<std::vector<int>> a;
    a.push_back(std::vector<int>({100, 1, 100}));
    a.push_back(std::vector<int>({1000, 1, 1000}));
    a.push_back(std::vector<int>({10000, 10, 1000}));
    a.push_back(std::vector<int>({100000, 100, 1000}));
    a.push_back(std::vector<int>({1000000, 1000, 1000}));

    std::ofstream output;
    output.open("add_benchmark.csv", std::ios_base::trunc);
    output << "Size"<< "," << "GPU" << "," << "CPU" << std::endl;
    for(auto elem: a) {
        auto res = sum_benchmark(elem[0], elem[1], elem[2]);
        output << elem[0] << "," << res.first << "," << res.second << std::endl;
    }
    output.close();
}

void matrix_bench(){
    std::vector<int> tests{64, 300, 500, 800, 1000};
    std::ofstream output;
    output.open("matrix_benchmark.csv", std::ios_base::trunc);
    output << "Size"<< "," << "GPU" << "," << "CPU" << std::endl;
    for(auto t: tests){
        auto res = matrix_benchmark(t);
        output << t << "," << res.first << "," << res.second << std::endl;
    }
    output.close();
}

int main() {
    matrix_bench();
    sum_bench();
    return 0;
}
